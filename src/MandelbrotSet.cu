#include "hip/hip_runtime.h"
#include "MandelbrotSet.cuh"

__global__ void computeKernel(uint8_t *data, int width, int height, double x_start, double x_finish, double y_start, double y_finish, vec3 *colortable_device);
/* 1. basic algorithm */
__global__ void basic_kernel(uint8_t *data, 
                                uint width,
                                uint height, 
                                double x_start, 
                                double x_finish, 
                                double y_start, 
                                double y_finish,
                                int maxiter);

/* 2. escape time based algorithm */
__global__ void escapetime_kernel(uint8_t *data, 
                                uint width,
                                uint height, 
                                double x_start, 
                                double x_finish, 
                                double y_start, 
                                double y_finish,
                                int maxiter,
                                vec3* colortable,
                                int ncycle,
                                double stripe_s,
                                double stripe_sig,
                                double step_s,
                                double phi,
                                double theta,
                                double opacity,
                                double k_ambient,
                                double k_diffuse,
                                double k_specular,
                                double shininess);

__device__ void smooth_iter(hipDoubleComplex c, 
                            int maxiter, 
                            double stripe_s,
                            double stripe_sig,
                            double &niter,
                            double &stripe_a,
                            double &dem,
                            hipDoubleComplex &normal);

__device__ __forceinline__ void color_pixel(uint8_t *data, 
                                            double niter,
                                            double stripe_a,
                                            double step_s,
                                            double dem,
                                            hipDoubleComplex normal,
                                            vec3* colortable,
                                            int ncycle,
                                            double phi,
                                            double theta,
                                            double opacity,
                                            double k_ambient,
                                            double k_diffuse,
                                            double k_specular,
                                            double shininess);

__device__ __forceinline__ void overlay(double x, 
                                        double y,
                                        double gamma,
                                        double &output); 

__device__ __forceinline__ void blinn_phong_lighting(hipDoubleComplex normal, 
                                                    double phi,
                                                    double theta,
                                                    double opacity,
                                                    double k_ambient,
                                                    double k_diffuse,
                                                    double k_specular,
                                                    double shininess,
                                                    double &brightness);

MandelbrotSet::MandelbrotSet(int w, int h) : width(w), height(h)
{
    data_host.resize(width * height * 3);
    data_device = data_host;
    colortable_host = colormap();
    colortable_device = colortable_host;

    ncycle=sqrt(ncycle);
    light[0]=2*PI*light[0]/360.;
    light[1]=PI/2.*light[1]/90.;

}

MandelbrotSet::~MandelbrotSet()
{
}

std::vector<vec3> MandelbrotSet::colormap(vec3 theta, int color_size)
{
    double start = 0, finish = 1;
    double dx = (finish - start) / color_size;
    std::vector<vec3> colors(color_size + 1);
    // #pragma omp parallel for
    for (int i = 0; i <= color_size; ++i)
    {
        vec3 color;
        color = (vec3(dx * i) + theta) * 2.0f * PI;
        color.x = 0.5f + std::sin(color.x);
        color.y = 0.5f + std::sin(color.y);
        color.z = 0.5f + std::sin(color.z);
        colors[i] = color;
    }
    return colors;
}

void MandelbrotSet::update_colormap(vec3 theta) {
    colortable_host = colormap(theta);
    colortable_device = colortable_host;
}

int MandelbrotSet::calpixel(std::complex<double> c)
{
    int count = 0;
    std::complex<double> z = c;
    double tmp, lengthsq;
    int max = max_iterations;
    do
    {
        /*tmp=(z.real()*z.real()-z.imag()*z.imag())+c.real();
        z.imag(2.0f*z.real()*z.imag()+c.imag());
        z.real(tmp);*/
        z = z * z + c;
        lengthsq = std::norm(z);
        ++count;
    } while ((lengthsq < 4.0) && (count < max));
    return count;
}

#define TILE_WIDTH 32
void MandelbrotSet::compute(double x_start, double x_finish, double y_start, double y_finish)
{
    dim3 dimGrid(ceil((double)width / TILE_WIDTH), ceil((double)height / TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    uint8_t *dataptr = thrust::raw_pointer_cast(&data_device[0]);
    vec3 *colortableptr = thrust::raw_pointer_cast(&colortable_device[0]);
    computeKernel<<<dimGrid, dimBlock>>>(dataptr, width, height, x_start, x_finish, y_start, y_finish, colortableptr);
    data_host = data_device;
}

__global__ void computeKernel(uint8_t *data, int width, int height, double x_start, double x_finish, double y_start, double y_finish, vec3 *colortable_device)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    static const double esc_radius = 2;

    if ((col < width) && (row < height))
    {
        double dx = (x_finish - x_start) / (width - 1);
        double dy = (y_finish - y_start) / (height - 1);
        int offset = (width * row + col) * 3;
        int count = 0;
        hipDoubleComplex c{col * dx + x_start, row * dy + y_start};
        hipDoubleComplex z{0, 0};
        int max = 256;
        do
        {
            z = hipCadd(hipCmul(z, z), c);
            ++count;
        }while ((hipCabs(z) <= esc_radius) && (count < max));

        int color_index = count == max ? max : count + 1 - log(log2(hipCabs(z)));
        vec3 color = colortable_device[color_index];
        data[offset + 0] = uint8_t(color.x*255);
        data[offset + 1] = uint8_t(color.y*255);
        data[offset + 2] = uint8_t(color.z*255);
    }
}

void MandelbrotSet::basic_algorithm(double x_start, double x_finish, double y_start, double y_finish)
{
    uint S = width * height;
    uint8_t *dataptr = thrust::raw_pointer_cast(&data_device[0]);
    basic_kernel<<<ceil(S/512.), 512>>>(dataptr, width, height, x_start, x_finish, y_start, y_finish, maxiter);
    data_host = data_device;
}

void MandelbrotSet::escapetime_based_algorithm(double x_start, double x_finish, double y_start, double y_finish) {
    uint S = width * height;
    uint8_t *dataptr = thrust::raw_pointer_cast(&data_device[0]);
    vec3 *colortableptr = thrust::raw_pointer_cast(&colortable_device[0]);
    escapetime_kernel<<<ceil(S/512.), 512>>>(dataptr, width, height, x_start, x_finish, y_start, y_finish, maxiter, colortableptr, ncycle, stripe_s, stripe_sig, step_s, light[0], light[1], light[2], light[3], light[4], light[5], light[6]);
    hipDeviceSynchronize();
    data_host = data_device;
}

__global__ void basic_kernel(uint8_t *data, 
                                uint width,
                                uint height, 
                                double x_start, 
                                double x_finish, 
                                double y_start, 
                                double y_finish,
                                int maxiter){
    uint S = width * height;
    cuda_foreach_uint(x, 0, S) {
        uint row = x / width;
        uint col = x % width;
        double dx = (x_finish - x_start) / (width - 1);
        double dy = (y_finish - y_start) / (height - 1);
        int offset = (width * row + col) * 3;
        hipDoubleComplex z{0, 0};
        hipDoubleComplex c{col * dx + x_start, row * dy + y_start};
        data[offset] = 0;
        data[offset + 1] = 0;
        data[offset + 2] = 0;
        for (int i = 0; i < maxiter; ++i) {
            z = hipCadd(hipCmul(z, z), c);
            if (hipCabs(z) > 2) {
                data[offset] = 255;
                data[offset + 1] = 255;
                data[offset + 2] = 255;
                break;
            }
        }
    }
}

__global__ void escapetime_kernel(uint8_t *data, 
                                uint width,
                                uint height, 
                                double x_start, 
                                double x_finish, 
                                double y_start, 
                                double y_finish,
                                int maxiter,
                                vec3* colortable,
                                int ncycle,
                                double stripe_s,
                                double stripe_sig,
                                double step_s,
                                double phi,
                                double theta,
                                double opacity,
                                double k_ambient,
                                double k_diffuse,
                                double k_specular,
                                double shininess){
    uint S = width * height;
    cuda_foreach_uint(x, 0, S) {
        uint row = x / width;
        uint col = x % width;
        double dx = (x_finish - x_start) / (width - 1);
        double dy = (y_finish - y_start) / (height - 1);
        int offset = (width * row + col) * 3;
        hipDoubleComplex z{0, 0};
        hipDoubleComplex c{col * dx + x_start, row * dy + y_start};

        double niter, stripe_a, dem;
        hipDoubleComplex normal;
        smooth_iter(c, maxiter, stripe_s, stripe_sig, niter, stripe_a, dem, normal);
        if(niter > 0) {
            color_pixel(data+offset, niter, stripe_a, step_s, dem, normal, colortable, ncycle, phi, theta, opacity, k_ambient, k_diffuse, k_specular, shininess);
        }
        else{
            data[offset] = 0;
            data[offset + 1] = 0;
            data[offset + 2] = 0;
        }
    }
}
                            
__device__ void smooth_iter(hipDoubleComplex c, 
                            int maxiter, 
                            double stripe_s,
                            double stripe_sig,
                            double &niter,
                            double &stripe_a,
                            double &dem,
                            hipDoubleComplex &normal) {
    hipDoubleComplex z{0, 0};
    hipDoubleComplex dz{1, 0};
    hipDoubleComplex two{2, 2};
    hipDoubleComplex one{1, 0};

    double esc_radius = 1e5; 

    bool is_stripe = (stripe_s > 0) && (stripe_sig > 0);
    double stripe_tt;
    double modz;

    int n = 0;
    for(n = 0; n < maxiter; ++n) {
        dz = hipCadd(hipCmul(two, hipCmul(z, dz)), one);
        z = hipCadd(hipCmul(z, z), c);
        if(is_stripe) {
            stripe_tt = (sin(stripe_s*atan2(hipCimag(z), hipCreal(z)))+1) / 2.;
        }
        modz = hipCabs(z);
        if (modz > esc_radius) {
            double log_ratio = log(modz) / log(esc_radius);
            double smooth_i =  1 - log(log_ratio) / log(2.);
            if(is_stripe) {
                stripe_a = (stripe_a * (1 + smooth_i * (stripe_sig-1)) + stripe_tt * smooth_i * (1 - stripe_sig));
                stripe_a = stripe_a / (1 - pow(stripe_sig, n) * (1 + smooth_i * (stripe_sig-1)));
            }
            normal = hipCdiv(z, dz);
            dem = modz * log(modz) / hipCabs(dz) / 2;
            niter = n + smooth_i;
            break;
        }

        if (is_stripe) {
            stripe_a = stripe_a * stripe_sig + stripe_tt * (1 - stripe_sig);
        }
    }
    if(n == maxiter) {
        niter = 0;
        stripe_a = 0;
        dem = 0;
        normal = {0, 0};
    }
}

__device__ __forceinline__ void color_pixel(uint8_t *data, 
                                            double niter,
                                            double stripe_a,
                                            double step_s,
                                            double dem,
                                            hipDoubleComplex normal,
                                            vec3* colortable,
                                            int ncycle,
                                            double phi,
                                            double theta,
                                            double opacity,
                                            double k_ambient,
                                            double k_diffuse,
                                            double k_specular,
                                            double shininess) {
    int ncol = (colortable_size) - 1;
    double iter = (double)((int)sqrt(niter) % ncycle) / ncycle;
    int col_i = round(iter * ncol);

    double brightness;
    blinn_phong_lighting(normal, phi, theta, opacity, k_ambient, k_diffuse, k_specular, shininess, brightness);

    dem = -log(dem) / 12;
    dem = 1 / (1 + exp(-10 * ((2*dem-1)/2)));

    int nshader = 0;
    double shader = 0;

    if(stripe_a > 0) {
        nshader += 1;
        shader = shader + stripe_a;
    } 
    if(step_s > 0) {
        step_s = 1/step_s;
        col_i = round((iter - (int)iter % (int)step_s)*ncol);
        double x = (int)iter % (int)step_s / step_s;
        double light_step = 6*(1-pow(x,5)-pow(1-x,100))/10;
        step_s = step_s/8;
        x = (int)iter % (int)step_s / step_s;
        double light_step2 = 6*(1-pow(x,5)-(1-x,30))/10;
        double light_step_mixed;
        overlay(light_step2, light_step, 1, light_step_mixed);
        nshader += 1;
        shader = shader + light_step_mixed;
    }
    if(nshader > 0) {
        double light;
        shader = shader / nshader;
        overlay(brightness, shader, 1, light);
        brightness = light * (1-dem) + dem * brightness;
    }
    vec3& color = colortable[col_i];
    double r, g, b;
    overlay(color.x, brightness, 1, r);
    overlay(color.y, brightness, 1, g);
    overlay(color.z, brightness, 1, b);
    r = fmaxf(0, fminf(1, r));
    g = fmaxf(0, fminf(1, g));
    b = fmaxf(0, fminf(1, b));

    data[0] = uint8_t(r * 255);
    data[1] = uint8_t(g * 255);
    data[2] = uint8_t(b * 255);

}

__device__ __forceinline__ void overlay(double x, 
                                        double y,
                                        double gamma,
                                        double &output) {
    if(y < 0.5)
        output = 2 * x * y;
    else
        output = 1 - 2 * (1 - x) * (1 - y);
    output = output * gamma + x * (1 - gamma);

}

__device__ __forceinline__ void blinn_phong_lighting(hipDoubleComplex normal, 
                                                    double phi,
                                                    double theta,
                                                    double opacity,
                                                    double k_ambient,
                                                    double k_diffuse,
                                                    double k_specular,
                                                    double shininess,
                                                    double &brightness) {
    hipDoubleComplex modn{hipCabs(normal), hipCabs(normal)};                                                    
    normal = hipCdiv(normal, modn);
    // Diffuse
    double ldiff = hipCreal(normal)*cos(phi)*cos(theta) + hipCimag(normal)*sin(phi)*cos(theta) + sin(theta);
    ldiff = ldiff / (1 + sin(theta));

    // Specular
    double theta_half = (PI/2. + theta) / 2.;
    double lspec = hipCreal(normal)*cos(phi)*sin(theta_half) + hipCimag(normal)*sin(phi)*sin(theta_half) + cos(theta_half);
    lspec = lspec / (1 + cos(theta_half));
    lspec = pow(lspec, shininess);

    brightness = k_ambient + k_diffuse * ldiff + k_specular * lspec;
    brightness = brightness * opacity + (1 - opacity)/2.;

}